#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
using namespace std;
#define READ_INT(n) {char c; n = getchar_unlocked() - '0'; while((c = getchar_unlocked()) >= '0') n = (n << 3) + (n << 1) + c - '0';}
#include "timer.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

int n,m,*edg,*degree,*startNode,*endNode;
thrust::host_vector<thrust::pair<int,int> > stEdges;
int md,*dedg,*dstartNode,*dendNode,*dresult;
int threads_per_block = 1024,blocks_per_grid = 16;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


// use 64 bits for sort ?

__global__ void numTri(int m,int * __restrict__ edg,int * __restrict__ startNode,int * __restrict__ endNode,int * result) {
    int t = blockDim.x * blockIdx.x + threadIdx.x,ret = 0;
    int numThreads = gridDim.x * blockDim.x; 
    for(int i=t;i<m;i += numThreads) {
    	int u = edg[i],v = edg[m+i];
		int su = startNode[u],eu = endNode[u]; int sv = startNode[v],ev = endNode[v];
		if(su != -1 and sv != -1) {
			while(su <= eu and sv <= ev) {
				if(edg[su+m] == edg[sv+m]) {
					su++; sv++; ret++;
				}
				else if(edg[su+m] > edg[sv+m]) sv++;
				else su++;
			}
		}
    }
    result[t] = ret;
}

void setupDeviceMemory() {
	int sizeVer = n * sizeof(int),sizeEdg = 2*m * sizeof(int);
	int tem = threads_per_block * blocks_per_grid * sizeof(int);
	gpuErrchk(hipMalloc(&dedg,sizeEdg));
	gpuErrchk(hipMalloc(&dstartNode,sizeVer));
	gpuErrchk(hipMalloc(&dendNode,sizeVer));
	gpuErrchk(hipMalloc(&dresult,tem)); gpuErrchk(hipMemset(dresult,0,tem)); 
   	gpuErrchk(hipMemcpy(dedg,edg,sizeEdg,hipMemcpyHostToDevice));
   	gpuErrchk(hipMemcpy(dstartNode,startNode,sizeVer,hipMemcpyHostToDevice));
   	gpuErrchk(hipMemcpy(dendNode,endNode,sizeVer,hipMemcpyHostToDevice));
}

void freeDeviceMemory() {
	free(edg); free(degree); free(startNode); free(endNode);
	gpuErrchk(hipFree(dedg)); gpuErrchk(hipFree(dstartNode)); gpuErrchk(hipFree(dendNode)); gpuErrchk(hipFree(dresult));
}

int main() {
	READ_INT(n); READ_INT(m);
	int sizeVer = n * sizeof(int),sizeEdg = 2*m * sizeof(int);
	edg = (int *) malloc(sizeEdg);
	degree = (int *) malloc(sizeVer);
	startNode = (int *) malloc(sizeVer);
	endNode = (int *) malloc(sizeVer);
	for(int i=0;i<n;i++) {
		degree[i] = -1; startNode[i] = -1; endNode[i] = -1;
	}
	for(int i = 0 ; i < m ; i++) {
		int node1,node2;
		READ_INT(node1); READ_INT(node2);
		stEdges.push_back(thrust::make_pair(node1,node2));
		degree[node1]++; degree[node2]++;
	}
	for(int i = 0 ;i < stEdges.size(); i++) {
		if(degree[stEdges[i].first] > degree[stEdges[i].second]) {
			thrust::swap(stEdges[i].first,stEdges[i].second);
			swap(stEdges[i].first,stEdges[i].second);

		}
	}
	thrust::device_vector<thrust::pair<int,int> > dEdg = stEdges;
	thrust::sort(dEdg.begin(),dEdg.end());
	stEdges = dEdg;
	
	for(int i = 0 ;i < stEdges.size(); i++) {
		edg[i] = stEdges[i].first; edg[i+m] = stEdges[i].second;
		if(startNode[stEdges[i].first] == -1) startNode[stEdges[i].first] = i;
		endNode[stEdges[i].first] = i;
	}
	double start,finish;
	GET_TIME(start);
	setupDeviceMemory();
	numTri<<<blocks_per_grid,threads_per_block>>>(m,dedg,dstartNode,dendNode,dresult);
	hipDeviceSynchronize();
	thrust::device_ptr<int> dptr(dresult);
	int  result = thrust::reduce(dptr,dptr+(threads_per_block*blocks_per_grid));
	printf("%d\n",result);
	GET_TIME(finish);
	hipDeviceSynchronize();
	freeDeviceMemory();
	printf("Elapsed time = %e seconds\n",finish - start);
}